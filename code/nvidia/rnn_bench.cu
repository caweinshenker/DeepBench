#include <chrono>
#include <iomanip>
#include <memory>
#include <stdexcept>
#include <tuple>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include "tensor_16.h"
#include "cudnn_helper.h"

hipdnnHandle_t cudnn_handle;
hiprandGenerator_t curand_gen;


class cudnnDropout {
    std::shared_ptr<hipdnnDropoutDescriptor_t> dropout_desc_;
    std::shared_ptr<Tensor<uint8_t>> dropout_state_;

    struct DropoutDeleter {
        void operator()(hipdnnDropoutDescriptor_t * dropout_desc) {
            hipdnnDestroyDropoutDescriptor(*dropout_desc);
            delete dropout_desc;
        }
    };

    public:

    cudnnDropout(float dropout_percentage) : dropout_desc_(new hipdnnDropoutDescriptor_t,
                                                           DropoutDeleter()) {
        size_t dropoutStateSize;
        CHECK_CUDNN_ERROR(hipdnnCreateDropoutDescriptor(dropout_desc_.get()));
        CHECK_CUDNN_ERROR(hipdnnDropoutGetStatesSize(cudnn_handle, &dropoutStateSize));

        dropout_state_.reset(new Tensor<uint8_t>(std::vector<int>{static_cast<int>(dropoutStateSize), 1}));

        CHECK_CUDNN_ERROR(hipdnnSetDropoutDescriptor(*dropout_desc_,
                                                    cudnn_handle,
                                                    dropout_percentage,
                                                    dropout_state_->begin(),
                                                    dropoutStateSize,
                                                    0ULL) );
    }

    hipdnnDropoutDescriptor_t desc() const { return *dropout_desc_; }
};

class cudnnRNN {
    RNNDescriptor<float> rnn_desc_;
    FilterDescriptorNd<float> wDesc_;
    cudnnDropout dropout_;

    int time_steps_;

    TensorDescriptorNdArray<float> xDescArray_;
    TensorDescriptorNdArray<float> yDescArray_;
    TensorDescriptorNdArray<float> dxDescArray_;
    TensorDescriptorNdArray<float> dyDescArray_;

    TensorDescriptorNd<float> hx_desc_;
    TensorDescriptorNd<float> hy_desc_;
    TensorDescriptorNd<float> dhx_desc_;
    TensorDescriptorNd<float> dhy_desc_;
    TensorDescriptorNd<float> cx_desc_;
    TensorDescriptorNd<float> cy_desc_;
    TensorDescriptorNd<float> dcx_desc_;
    TensorDescriptorNd<float> dcy_desc_;

    size_t weight_size_;
    size_t workspace_size_;
    size_t train_size_;

    Tensor<float> weights_;
    Tensor<float> workspace_;
    Tensor<float> trainspace_;

    public:

    cudnnRNN(int hidden_size, int batch_size, int time_steps, const std::string& rnn_type) :
        dropout_(0.f), time_steps_(time_steps),
        xDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        yDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        dxDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        dyDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        hx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        hy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dhx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dhy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        cx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        cy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dcx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dcy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1})
        {


            rnn_desc_ = RNNDescriptor<float>(hidden_size,
                                             1,
                                             dropout_.desc(),
                                             HIPDNN_SKIP_INPUT,
                                             HIPDNN_UNIDIRECTIONAL,
                                             rnn_type);

            CHECK_CUDNN_ERROR( hipdnnGetRNNParamsSize(cudnn_handle,
                                                     rnn_desc_.desc(),
                                                     xDescArray_.ptr()[0],
                                                     &weight_size_,
                                                     HIPDNN_DATA_FLOAT) );

            weights_ = rand(std::vector<int>{static_cast<int>(weight_size_ / sizeof(float)), 1}, curand_gen);


            std::vector<int> dim = {weights_.size(), 1, 1};
            wDesc_ = FilterDescriptorNd<float>(HIPDNN_TENSOR_NCHW, dim);

            CHECK_CUDNN_ERROR( hipdnnGetRNNWorkspaceSize(cudnn_handle,
                                                        rnn_desc_.desc(),
                                                        time_steps,
                                                        xDescArray_.ptr(),
                                                        &workspace_size_) );

            workspace_ = zeros(std::vector<int>{static_cast<int>(workspace_size_ / sizeof(float)), 1});

            CHECK_CUDNN_ERROR( hipdnnGetRNNTrainingReserveSize(cudnn_handle,
                                                              rnn_desc_.desc(),
                                                              time_steps,
                                                              xDescArray_.ptr(),
                                                              &train_size_) );
            trainspace_ = zeros(std::vector<int>{static_cast<int>(train_size_ / sizeof(float)), 1});
        }
        void forward(Tensor<float> x, Tensor<float> hx, Tensor<float> cx,
                     Tensor<float> y, Tensor<float> hy, Tensor<float> cy) {
            CHECK_CUDNN_ERROR( hipdnnRNNForwardTraining(cudnn_handle,
                                                       rnn_desc_.desc(),
                                                       time_steps_,
                                                       xDescArray_.ptr(),
                                                       (void *)x.begin(),
                                                       hx_desc_.desc(),
                                                       (void *)hx.begin(),
                                                       cx_desc_.desc(),
                                                       (void *)cx.begin(),
                                                       wDesc_.desc(),
                                                       (void *)weights_.begin(),
                                                       yDescArray_.ptr(),
                                                       (void *)y.begin(),
                                                       hy_desc_.desc(),
                                                       (void *)hy.begin(),
                                                       cy_desc_.desc(),
                                                       (void *)cy.begin(),
                                                       (void *)workspace_.begin(),
                                                       workspace_size_,
                                                       (void *)trainspace_.begin(),
                                                       train_size_) );
        }
        void backward_data(Tensor<float> y, Tensor<float> dy, Tensor<float> dhy,
                           Tensor<float> dcy, Tensor<float> hx, Tensor<float> cx,
                           Tensor<float> dx, Tensor<float> dhx, Tensor<float> dcx) {
            CHECK_CUDNN_ERROR( hipdnnRNNBackwardData(cudnn_handle,
                                                    rnn_desc_.desc(),
                                                    time_steps_,
                                                    yDescArray_.ptr(),
                                                    (void *)y.begin(),
                                                    dyDescArray_.ptr(),
                                                    (void *)dy.begin(),
                                                    dhy_desc_.desc(),
                                                    (void *)dhy.begin(),
                                                    dcy_desc_.desc(),
                                                    (void *)dcy.begin(),
                                                    wDesc_.desc(),
                                                    (void *)weights_.begin(),
                                                    hx_desc_.desc(),
                                                    (void *)hx.begin(),
                                                    cx_desc_.desc(),
                                                    (void *)cx.begin(),
                                                    dxDescArray_.ptr(),
                                                    (void *)dx.begin(),
                                                    dhx_desc_.desc(),
                                                    (void *)dhx.begin(),
                                                    dcx_desc_.desc(),
                                                    (void *)dcx.begin(),
                                                    (void *)workspace_.begin(),
                                                    workspace_size_,
                                                    (void *)trainspace_.begin(),
                                                    train_size_) );
        }
};

std::tuple<int, int> time_rnn(int hidden_size,
                              int batch_size,
                              int time_steps,
                              const std::string& type) {

    cudnnRNN rnn(hidden_size, batch_size, time_steps, type);

    auto x  = rand({hidden_size, batch_size * time_steps}, curand_gen);
    auto y  = rand({hidden_size, batch_size * time_steps}, curand_gen);
    auto dx = rand({hidden_size, batch_size * time_steps}, curand_gen);
    auto dy = rand({hidden_size, batch_size * time_steps}, curand_gen);

    auto hx = rand({hidden_size, batch_size}, curand_gen);
    auto hy = rand({hidden_size, batch_size}, curand_gen);
    auto cx = rand({hidden_size, batch_size}, curand_gen);
    auto cy = rand({hidden_size, batch_size}, curand_gen);
    auto dhx = rand({hidden_size, batch_size}, curand_gen);
    auto dhy = rand({hidden_size, batch_size}, curand_gen);
    auto dcx = rand({hidden_size, batch_size}, curand_gen);
    auto dcy = rand({hidden_size, batch_size}, curand_gen);

    int numRepeats = 100;

    //Warm up
    rnn.forward(x, hx, cx, y, hy, cy);

    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i) {
        rnn.forward(x, hx, cx, y, hy, cy);
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();

    auto forward_time = std::chrono::duration<double, std::micro>(end - start).count() / numRepeats;

    //Warm up
    rnn.backward_data(y, dy, dhy, dcy,
                      hx, cx, dx, dhx, dcx);

    hipDeviceSynchronize();

    start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i) {
        rnn.backward_data(y, dy, dhy, dcy,
                          hx, cx, dx, dhx, dcx);
    }
    hipDeviceSynchronize();

    end = std::chrono::steady_clock::now();
    auto backward_time = std::chrono::duration<double, std::micro>(end - start).count() / numRepeats;

    return std::make_tuple(static_cast<int>(forward_time),
                           static_cast<int>(backward_time));

}

int main(int argc, char **argv) {
    hipFree(0);
    CHECK_CUDNN_ERROR( hipdnnCreate(&cudnn_handle) );

    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    std::vector<std::tuple<int, int, int, bool>> problems  = {
	std::make_tuple(1760, 16, 50, false),
        std::make_tuple(1760, 32, 50, false),
        std::make_tuple(1760, 64, 50, false),
        std::make_tuple(1760, 128, 50, false),
        std::make_tuple(2048, 16, 50, false),
        std::make_tuple(2048, 32, 50, false),
        std::make_tuple(2048, 64, 50, false),
        std::make_tuple(2048, 128, 50, false),
        std::make_tuple(2560, 16, 50, false),
        std::make_tuple(2560, 32, 50, false),
        std::make_tuple(2560, 64, 50, false),
        std::make_tuple(2560, 128, 50, false),
        std::make_tuple(512, 16, 25, true),
        std::make_tuple(512, 32, 25, true),
        std::make_tuple(512, 64, 25, true),
        std::make_tuple(512, 128, 25, true),
        std::make_tuple(1024, 16, 25, true),
        std::make_tuple(1024, 32, 25, true),
        std::make_tuple(1024, 64, 25, true),
        std::make_tuple(1024, 128, 25, true),
        std::make_tuple(2048, 16, 25, true),
        std::make_tuple(2048, 32, 25, true),
        std::make_tuple(2048, 64, 25, true),
        std::make_tuple(2048, 128, 25, true),
        std::make_tuple(4096, 16, 25, true),
        std::make_tuple(4096, 32, 25, true),
        std::make_tuple(4096, 64, 25, true),
        std::make_tuple(4096, 128, 25, true)
    };

    std::cout << std::setw(30) << "Times" << std::endl;
    std::cout << std::setfill('-') << std::setw(88) << "-" << std::endl;
    std::cout << std::setfill(' ');
    std::cout << "    type    hidden   N     timesteps      fwd_time (usec)   bwd_time (usec)" << std::endl;
    for (const auto &problem : problems) {
        int hidden_state, batch_size, time_steps;
        bool lstm;
        std::tie(hidden_state, batch_size, time_steps, lstm) = problem;
        std::string type = lstm ? "lstm" : "vanilla";

        std::cout << std::setw(8) << type;
        std::cout << std::setw(8) << hidden_state;
        std::cout << std::setw(8) << batch_size;
        std::cout << std::setw(8) << 25;
        int fwd_time, bwd_time;
        std::tie(fwd_time, bwd_time) = time_rnn(hidden_state,
                                                batch_size,
                                                time_steps,
                                                type);
        std::cout << std::setw(18) << fwd_time;
        std::cout << std::setw(18) << bwd_time;
        std::cout << std::endl;
    }

    hipdnnDestroy(cudnn_handle);
    hiprandDestroyGenerator(curand_gen);

    return 0;
}
